#include <iostream>
#include <opencv2/opencv.hpp>
#include "CudaKernel.h"

using namespace cv;
using namespace std;

int main(int argc, char** argv) 
{
    IplImage* image;

    image = cvLoadImage("4555472_460s.jpg", CV_LOAD_IMAGE_GRAYSCALE);

    if(!image )
    {
        cout << "Could not open or find the image" << std::endl;
        return -1;
    }


    IplImage* image2 = cvCreateImage(cvGetSize(image),IPL_DEPTH_32F,image->nChannels);
    IplImage* image3 = cvCreateImage(cvGetSize(image),IPL_DEPTH_32F,image->nChannels);

    //Convert the input image to float
    cvConvert(image,image3);

    float *output = (float*)image2->imageData;
    float *input =  (float*)image3->imageData;

    kernelcall(input, output, image->width,image->height, image3->widthStep);

    //Normalize the output values from 0.0 to 1.0
    cvScale(image2,image2,1.0/255.0);

    cvShowImage("Original Image", image );
    cvShowImage("Sobeled Image", image2);
    cvWaitKey(0);
    return 0;
}