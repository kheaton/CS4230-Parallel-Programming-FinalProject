
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include "string.h"


#define DEFAULT_THRESHOLD  4000

#define DEFAULT_FILENAME "BWstop-sign.ppm"

unsigned int *read_ppm( char *filename, int * xsize, int * ysize, int *maxval ){
  
  if ( !filename || filename[0] == '\0') {
    fprintf(stderr, "read_ppm but no file name\n");
    return NULL;  // fail
  }

  FILE *fp;

  fprintf(stderr, "read_ppm( %s )\n", filename);
  fp = fopen( filename, "rb");
  if (!fp) 
    {
      fprintf(stderr, "read_ppm()    ERROR  file '%s' cannot be opened for reading\n", filename);
      return NULL; // fail 

    }

  char chars[1024];
  //int num = read(fd, chars, 1000);
  int num = fread(chars, sizeof(char), 1000, fp);

  if (chars[0] != 'P' || chars[1] != '6') 
    {
      fprintf(stderr, "Texture::Texture()    ERROR  file '%s' does not start with \"P6\"  I am expecting a binary PPM file\n", filename);
      return NULL;
    }

  unsigned int width, height, maxvalue;


  char *ptr = chars+3; // P 6 newline
  if (*ptr == '#') // comment line! 
    {
      ptr = 1 + strstr(ptr, "\n");
    }

  num = sscanf(ptr, "%d\n%d\n%d",  &width, &height, &maxvalue);
  fprintf(stderr, "read %d things   width %d  height %d  maxval %d\n", num, width, height, maxvalue);  
  *xsize = width;
  *ysize = height;
  *maxval = maxvalue;
  
  unsigned int *pic = (unsigned int *)malloc( width * height * sizeof(unsigned int));
  if (!pic) {
    fprintf(stderr, "read_ppm()  unable to allocate %d x %d unsigned ints for the picture\n", width, height);
    return NULL; // fail but return
  }

  // allocate buffer to read the rest of the file into
  int bufsize =  3 * width * height * sizeof(unsigned char);
  if ((*maxval) > 255) bufsize *= 2;
  unsigned char *buf = (unsigned char *)malloc( bufsize );
  if (!buf) {
    fprintf(stderr, "read_ppm()  unable to allocate %d bytes of read buffer\n", bufsize);
    return NULL; // fail but return
  }

  // TODO really read
  char duh[80];
  char *line = chars;

  // find the start of the pixel data.   no doubt stupid
  sprintf(duh, "%d\0", *xsize);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d\0", *ysize);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d\0", *maxval);
  line = strstr(line, duh);


  fprintf(stderr, "%s found at offset %d\n", duh, line - chars);
  line += strlen(duh) + 1;

  long offset = line - chars;
  //lseek(fd, offset, SEEK_SET); // move to the correct offset
  fseek(fp, offset, SEEK_SET); // move to the correct offset
  //long numread = read(fd, buf, bufsize);
  long numread = fread(buf, sizeof(char), bufsize, fp);
  fprintf(stderr, "Texture %s   read %ld of %ld bytes\n", filename, numread, bufsize); 

  fclose(fp);


  int pixels = (*xsize) * (*ysize);
  for (int i=0; i<pixels; i++) pic[i] = (int) buf[3*i];  // red channel

 

  return pic; // success
}

void write_ppm( char *filename, int xsize, int ysize, int maxval, int *pic) {
  FILE *fp;
  //int x,y;
  
  fp = fopen(filename, "w");
  if (!fp) 
    {
      fprintf(stderr, "FAILED TO OPEN FILE '%s' for writing\n");
      exit(-1); 
    }
  
  fprintf(fp, "P6\n"); 
  fprintf(fp,"%d %d\n%d\n", xsize, ysize, maxval);
  
  int numpix = xsize * ysize;
  for (int i=0; i<numpix; i++) {
    unsigned char uc = (unsigned char) pic[i];
    fprintf(fp, "%c%c%c", uc, uc, uc); 
  }
  fclose(fp);

}


int main( int argc, char **argv )
{
    int thresh = DEFAULT_THRESHOLD;
    int number_of_files = 20000;//21312;
    //filename = strdup( DEFAULT_FILENAME);
    hipEvent_t start_event, stop_event;
	float seq_time_gpu;

	if(argc > 1) {
        number_of_files = atoi(argv[1]);
	}

	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	hipEventRecord(start_event, 0);

    for(int k = 1; k <= number_of_files; k++) {

		char *in_filename = (char*)malloc(36 * sizeof(char));
		char *out_filename = (char*)malloc(36 * sizeof(char));

		sprintf(in_filename, "./sintel/sintel%03d.ppm", k);
		sprintf(out_filename, "./sintel-sobel-seq/sintel-sobel%03d.ppm", k);

		int xsize, ysize, maxval;
		unsigned int *pic = read_ppm( in_filename, &xsize, &ysize, &maxval ); 

		int numbytes =  xsize * ysize * 3 * sizeof( int );
		int *result = (int *) malloc( numbytes );
		if (!result) { 
			fprintf(stderr, "sobel() unable to malloc %d bytes\n", numbytes);
			exit(-1); // fail
		}

		int i, j, magnitude, sum1, sum2; 
		int *out = result;

		for (int col=0; col<ysize; col++) {
			for (int row=0; row<xsize; row++) { 
			*out++ = 0; 
			}
		}

		for (i = 1;  i < ysize - 1; i++) {
			for (j = 1; j < xsize -1; j++) {
				int offset = i*xsize + j;

				sum1 =  pic[ xsize * (i-1) + j+1 ] -     pic[ xsize*(i-1) + j-1 ] 
					+ 2 * pic[ xsize * (i)   + j+1 ] - 2 * pic[ xsize*(i)   + j-1 ]
					+     pic[ xsize * (i+1) + j+1 ] -     pic[ xsize*(i+1) + j-1 ];
				
				sum2 = pic[ xsize * (i-1) + j-1 ] + 2 * pic[ xsize * (i-1) + j ]  + pic[ xsize * (i-1) + j+1 ]
					- pic[xsize * (i+1) + j-1 ] - 2 * pic[ xsize * (i+1) + j ] - pic[ xsize * (i+1) + j+1 ];
				
				magnitude =  sum1*sum1 + sum2*sum2;

				if (magnitude > thresh)
					result[offset] = 255;
				else 
					result[offset] = 0;
			}
		}

		write_ppm( out_filename, xsize, ysize, 255, result);
	}

	hipEventRecord(stop_event, 0);

	hipEventSynchronize(stop_event);
	hipEventElapsedTime(&seq_time_gpu,start_event, stop_event); 

	printf("Sequential Time: %.2f msec\n", seq_time_gpu);

	fprintf(stderr, "sobel done\n"); 
}